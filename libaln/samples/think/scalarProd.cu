#include "hip/hip_runtime.h"
/* This code is partly from NVIDIA CUDA TOOLKIT.
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample calculates scalar products of a
 * given set of input vector pairs
 */



#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


void callCUDA();
void initGPU();
void execGPU();
void CheckResult();

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C"
void scalarProdCPU(
    float *h_C,
    float *h_A,
    float *h_B,
    int vectorN,
    int elementN
);



///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cuh"



////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
//float RandFloat(float low, float high)
//{
//    float t = (float)rand() / (float)RAND_MAX;
//    return (1.0f - t) * low + t * high;
//}



///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////

//Total number of input vector pairs; arbitrary
//Number of elements per vector; arbitrary,
//but strongly preferred to be a multiple of warp size
//to meet memory coalescing constraints

extern int ELEMENT_N;
extern int nNumberLFNs;
#define VECTOR_N nNumberLFNs
//Total number of data elements
const int    DATA_N = VECTOR_N * ELEMENT_N;

const int   DATA_SZ = DATA_N * sizeof(float);
const int RESULT_SZ = VECTOR_N  * sizeof(float);
float* h_C_CPU, * h_C_GPU;
float* d_A, * d_B, * d_C;

void callCUDA()
{
}

///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
void initGPU()
{
    int i;

    printf("%s Starting...\n\n", argv[0]);

    // use command-line specified CUDA device, otherwise use device with highest Gflops/s
    findCudaDevice(argc, (const char**)argv);

   

    printf("Initializing data...\n");
    printf("...allocating CPU memory.\n");
    // h_A = (float*)malloc(DATA_SZ); // these two are allocated in main, and are of maximal size
    //h_B = (float*)malloc(DATA_SZ);
    h_C_CPU = (float*)malloc(RESULT_SZ);
    h_C_GPU = (float*)malloc(RESULT_SZ);

    printf("...allocating GPU memory.\n");
    checkCudaErrors(hipMalloc((void**)&d_A, DATA_SZ));
    checkCudaErrors(hipMalloc((void**)&d_B, DATA_SZ));
    checkCudaErrors(hipMalloc((void**)&d_C, RESULT_SZ));

    execGPU();
    // We check inside a subroutine
    // CheckResult(h_C_CPU, h_C_GPU, h_A, h_B, VECTOR_N, ELEMENT_N);
    CheckResult();
    free(h_C_GPU);
    free(h_C_CPU);
    free(h_B);
    free(h_A);


    printf("Shutting down...\n");
    checkCudaErrors(hipFree(d_C));
    checkCudaErrors(hipFree(d_B));
    checkCudaErrors(hipFree(d_A));

}

void execGPU()
{
    StopWatchInterface* hTimer = NULL;
    sdkCreateTimer(&hTimer);
    printf("...copying input data to GPU mem.\n");
    //Copy options data to GPU memory for further processing
    checkCudaErrors(hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice));
    printf("Data init done.\n");


    printf("Executing GPU kernel...\n");
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&hTimer);
    sdkStartTimer(&hTimer);
    scalarProdGPU << <128, 256 >> > (d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
    getLastCudaError("scalarProdGPU() execution failed\n");
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&hTimer);
    printf("GPU time: %f msecs.\n", sdkGetTimerValue(&hTimer));
    sdkDeleteTimer(&hTimer);

    printf("Reading back GPU result...\n");
    //Read back GPU results to compare them to CPU results
    checkCudaErrors(hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost));
}


void CheckResult()
{
    double delta, ref, sum_delta, sum_ref, L1norm;

    printf("Checking GPU results...\n");
    printf("..running CPU scalar product calculation\n");
    scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);

    printf("...comparing the results\n");
    //Calculate max absolute difference and L1 distance
    //between CPU and GPU results
    sum_delta = 0;
    sum_ref   = 0;

    for (int i = 0; i < VECTOR_N; i++)
    {
        delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
        ref   = h_C_CPU[i];
        sum_delta += delta;
        sum_ref   += ref;
    }

    L1norm = sum_delta / sum_ref;



    printf("L1 error: %E\n", L1norm);
    printf((L1norm < 1e-6) ? "Test passed\n" : "Test failed!\n");
    exit(L1norm < 1e-6 ? EXIT_SUCCESS : EXIT_FAILURE);
}
